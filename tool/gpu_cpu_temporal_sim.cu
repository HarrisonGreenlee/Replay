#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h> /* for strcmp */
#include <time.h>

#define EIGEN_USE_THREADS
#include <Eigen/Dense>
#include <Eigen/Sparse>

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hipsparse.h>

extern "C" {
#include "intervaldb.h" /* for IntervalIterator, find_intervals, etc. */
#include "temporal_contact_matrix.h" /* <-- Our TCM library header */
}

/* ----------------------------------------------------------------------------
 * Simulation parameters
 * ----------------------------------------------------------------------------*/

static int num_nodes;
static int gM = 1000;             // Default number of parallel simulations
static float gStepSize = 3600.0f; // Default step size
static int gIterations = 42;      // Default number of iterations
static int gEigenThreads = -1;    // parallel threads for Eigen, default to max

FILE *summary_fp = NULL;
FILE *node_fp = NULL;

// Disease-state boundaries
#define SUSCEPTIBLE 0
// static int   gUpperRange             = 7200;
// static int   gMediumRange            = 3600;   // no resistance
// static int   gLowerRange             = 0;
static int gUpperRange;
static int gMediumRange;
static int gLowerRange;
// Duration parameters (user-facing, in seconds)
static int gExposedDuration;
static int gInfectiousDuration;
static int gResistantDuration;

static float gInitialInfectedProb; // Probability an individual starts infected
static float gSusceptibleInfectProb; // chance to get infected if exposed

static int64_t gGlobalStartTime = 946713600; // Jan 1, 2000
static int64_t gStaticNetworkDuration = 3600;// 1 hour

// Simple CUDA check macro
#define CUDA_CHECK(call)                                                       \
  do {                                                                         \
    hipError_t err = (call);                                                  \
    if (err != hipSuccess) {                                                  \
      fprintf(stderr, "CUDA error at %s:%d - %s\n", __FILE__, __LINE__,        \
              hipGetErrorString(err));                                        \
      exit(EXIT_FAILURE);                                                      \
    }                                                                          \
  } while (0)

/* ----------------------------------------------------------------------------
 * Device kernels
 * ----------------------------------------------------------------------------*/

__global__ void initialize_countdown_vector(int *countdown_vector,
                                            int totalSize, float infected_prob,
                                            float step_size, unsigned long seed,
                                            int medium_range,
                                            int susceptible_value) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < totalSize) {
    hiprandState state;
    hiprand_init(clock64() + idx + seed, 0, 0, &state);
    float rand_val = hiprand_uniform(&state);

    if (rand_val < infected_prob) {
      countdown_vector[idx] = medium_range + step_size;
    } else {
      countdown_vector[idx] = susceptible_value;
    }
  }
}

__global__ void generate_infectious_vector(const int *countdown_vector,
                                           float *infectious_vector,
                                           int n, // num_nodes
                                           int m, // gM
                                           int lower_range, int medium_range) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int total = n * m;
  if (idx < total) {
    // Convert idx -> (row, col) in col-major
    int row = idx % n;
    int col = idx / n;

    int c = countdown_vector[row + col * n];
    if (c > lower_range && c <= medium_range) {
      infectious_vector[row + col * n] = 1.0f;
    } else {
      infectious_vector[row + col * n] = 0.0f;
    }
  }
}

__global__ void compute_infection_probability(float *exposure_matrix,
                                              const int *countdown_vector,
                                              int n, int m, float step_size,
                                              float infect_prob,
                                              int susceptible_value) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int total = n * m;
  if (idx < total) {
    int row = idx % n;
    int col = idx / n;

    if (countdown_vector[row + col * n] == susceptible_value) {
      float log_base = logf(1.0f - infect_prob);
      //  we do not normalize wrt step size anymore - we now normalize wrt a
      //  standard "exposure hour". Just easier to work with. float e =
      //  exposure_matrix[row + col*n] / step_size;
      float e = exposure_matrix[row + col * n] / 3600.0f;
      exposure_matrix[row + col * n] = 1.0f - expf(log_base * e);
    } else {
      exposure_matrix[row + col * n] = 0.0f;
    }
  }
}

__global__ void monte_carlo_simulation(const float *prob_matrix,
                                       int *countdown_vector, int n, int m,
                                       unsigned long seed, float step_size,
                                       int upper_range, int susceptible_value) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int total = n * m;
  if (idx < total) {
    int row = idx % n;
    int col = idx / n;

    if (countdown_vector[row + col * n] == susceptible_value) {
      hiprandState state;
      hiprand_init(clock64() + idx + seed, 0, 0, &state);
      float rand_val = hiprand_uniform(&state);

      float infectionProb = prob_matrix[row + col * n];
      if (infectionProb > rand_val) {
        // New infection => set to incubation
        countdown_vector[row + col * n] = upper_range + step_size;
      }
    }
  }
}

__global__ void update_countdown_vector(int *countdown_vector, int n, int m,
                                        float step_size) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int total = n * m;
  if (idx < total) {
    int row = idx % n;
    int col = idx / n;

    if (countdown_vector[row + col * n] > 0) {
      countdown_vector[row + col * n] -= step_size;
    }
    if (countdown_vector[row + col * n] < 0) {
      countdown_vector[row + col * n] = SUSCEPTIBLE;
    }
  }
}

/* ----------------------------------------------------------------------------
 * Utility to print first few "rows" (meaning col=0..X in col-major)
 * ----------------------------------------------------------------------------*/
void print_status_counts(const int *h_countdown_vector, int n, int m,
                         int rowsToPrint) {
  for (int i = 0; i < rowsToPrint; i++) {
    int incubating = 0, infectious = 0, resistant = 0, susceptible = 0;
    for (int j = 0; j < n; j++) {
      int offset = j + i * n; // row=j, col=i
      int status = h_countdown_vector[offset];

      if (status == SUSCEPTIBLE)
        susceptible++;
      else if (status <= gLowerRange && status > 0)
        resistant++;
      else if (status <= gMediumRange)
        infectious++;
      else if (status > gMediumRange)
        incubating++;
    }
    printf("Row %d: Exposed=%d, Infectious=%d, Resistant=%d, Susceptible=%d\n",
           i, incubating, infectious, resistant, susceptible);
  }
}

/* ----------------------------------------------------------------------------
 * Build a CSR adjacency from the intervals that overlap [start_ts, end_ts)
 * We collect edges in a simple array, then sort them by row -> col, then
 * populate the CSR arrays.
 * ----------------------------------------------------------------------------*/
typedef struct {
  int row;
  int col;
  float weight;
} EdgeListItem;

/* Compare function for qsort (by row, then col) */
static int cmp_edgelist(const void *a, const void *b) {
  const EdgeListItem *ea = (const EdgeListItem *)a;
  const EdgeListItem *eb = (const EdgeListItem *)b;
  if (ea->row < eb->row)
    return -1;
  if (ea->row > eb->row)
    return 1;
  /* same row => compare col */
  if (ea->col < eb->col)
    return -1;
  if (ea->col > eb->col)
    return 1;
  return 0;
}

/*
 * build_csr_from_intervals:
 *   - 'hits' is the list of intervals that overlap [start_ts, end_ts).
 *   - 'n_hits' is how many we have.
 *   - If we treat them as *undirected*, we add both (src, tgt) and (tgt, src).
 *   - 'row' = src, 'col' = tgt, weight = overlap_duration.
 */
static void build_csr_from_intervals(IntervalMap *hits, int n_hits, int n_nodes,
                                     int **csrRowPtr, int **csrColInd,
                                     float **csrVal, size_t *p_nnz,
                                     int64_t start_ts, int64_t end_ts) {

  // --- Early exit if no intervals (avoid malloc(0) / invalid deref) ---
  if (n_hits <= 0) {
    *p_nnz = 0;
    *csrRowPtr = (int *)calloc((size_t)(n_nodes + 1), sizeof(int));
    *csrColInd = NULL;
    *csrVal = NULL;
    if (!*csrRowPtr) {
      fprintf(stderr, "Out of memory allocating empty CSR row pointer.\n");
      exit(1);
    }
    return;
  }

  typedef struct {
    int row;
    int col;
    float weight;
  } EdgeListItem;

  EdgeListItem *edgelist =
      (EdgeListItem *)malloc(sizeof(EdgeListItem) * (size_t)2 * n_hits);
  if (!edgelist) {
    fprintf(stderr, "Out of memory building edgelist.\n");
    exit(1);
  }

  int ecount = 0;

  for (int i = 0; i < n_hits; i++) {
    int64_t s = hits[i].start;
    int64_t e = hits[i].end;
    int src_idx, tgt_idx;
    unpack_node_pair(hits[i].target_id, &src_idx, &tgt_idx);

    int64_t overlap_start = (s > start_ts) ? s : start_ts;
    int64_t overlap_end = (e < end_ts) ? e : end_ts;
    if (overlap_end > overlap_start) {
      float overlap_duration = (float)(overlap_end - overlap_start);

      edgelist[ecount].row = src_idx;
      edgelist[ecount].col = tgt_idx;
      edgelist[ecount].weight = overlap_duration;
      ++ecount;

      edgelist[ecount].row = tgt_idx;
      edgelist[ecount].col = src_idx;
      edgelist[ecount].weight = overlap_duration;
      ++ecount;
    }
  }

  /* Sort by (row, col) */
  qsort(edgelist, ecount, sizeof(EdgeListItem), cmp_edgelist);

  /* Merge duplicates by summing weights */
  int agg = 0;
  for (int i = 1; i < ecount; i++) {
    if (edgelist[i].row == edgelist[agg].row &&
        edgelist[i].col == edgelist[agg].col) {
      edgelist[agg].weight += edgelist[i].weight;
    } else {
      ++agg;
      edgelist[agg] = edgelist[i];
    }
  }
  ecount = (ecount > 0) ? (agg + 1) : 0;

  *p_nnz = (size_t)ecount;
  *csrRowPtr = (int *)malloc((size_t)(n_nodes + 1) * sizeof(int));
  *csrColInd = (int *)malloc((size_t)(ecount) * sizeof(int));
  *csrVal = (float *)malloc((size_t)(ecount) * sizeof(float));

  if (!(*csrRowPtr) || !(*csrColInd) || !(*csrVal)) {
    fprintf(stderr, "Out of memory allocating CSR arrays.\n");
    free(edgelist);
    exit(1);
  }

  for (int i = 0; i <= n_nodes; i++) {
    (*csrRowPtr)[i] = 0;
  }

  for (int i = 0; i < ecount; i++) {
    int r = edgelist[i].row;
    (*csrRowPtr)[r + 1] += 1;
  }

  for (int i = 0; i < n_nodes; i++) {
    (*csrRowPtr)[i + 1] += (*csrRowPtr)[i];
  }

  int *rowPosition = (int *)malloc((size_t)n_nodes * sizeof(int));
  for (int i = 0; i < n_nodes; i++) {
    rowPosition[i] = (*csrRowPtr)[i];
  }

  for (int i = 0; i < ecount; i++) {
    int r = edgelist[i].row;
    int pos = rowPosition[r];
    (*csrColInd)[pos] = edgelist[i].col;
    (*csrVal)[pos] = edgelist[i].weight;
    rowPosition[r]++;
  }

  free(rowPosition);
  free(edgelist);
}

/* ----------------------------------------------------------------------------
 * CPU FALLBACK FUNCTIONS (newly added):
 * We replicate the logic of the CUDA kernels and SpMM using the host CPU.
 * These are only called if --cpu-only is specified.
 * ----------------------------------------------------------------------------*/

/*
 * CPUInitializeCountdownVector
 *  - Replicates initialize_countdown_vector kernel
 */
void CPUInitializeCountdownVector(int *countdown_vector, int totalSize,
                                  float infected_prob, float step_size) {
  // Simple seed using time(NULL), but we do it outside to keep it consistent
  srand((unsigned)time(NULL));
  for (int idx = 0; idx < totalSize; idx++) {
    float rand_val = (float)rand() / (float)RAND_MAX;
    if (rand_val < infected_prob) {
      countdown_vector[idx] = gMediumRange + step_size;
    } else {
      countdown_vector[idx] = SUSCEPTIBLE;
    }
  }
}

/*
 * CPUGenerateInfectiousVector
 *   - Replicates generate_infectious_vector kernel
 *   - NxM stored in column-major => element at [row + col*n].
 */
void CPUGenerateInfectiousVector(const int *countdown_vector,
                                 float *infectious_vector, int n, int m) {
  int total = n * m;
  for (int idx = 0; idx < total; idx++) {
    int row = idx % n;
    int col = idx / n;
    int c = countdown_vector[row + col * n];
    if (c > gLowerRange && c <= gMediumRange) {
      infectious_vector[idx] = 1.0f;
    } else {
      infectious_vector[idx] = 0.0f;
    }
  }
}

/*
 * CPUComputeSpMM
 *   - Replicates A * B => C, where A is CSR of size NxN,
 *     B is NxM (col-major),
 *     C is NxM (col-major).
 *   - alpha=1.0f, beta=0.0f
 */
void CPUComputeSpMM(const int *csrRowPtr, const int *csrColInd,
                    const float *csrVal, size_t nnz_count,
                    int n,                          // NxN
                    const float *infectious_vector, // NxM
                    float *exposure_matrix,         // NxM
                    int m) {
  using namespace Eigen;

  // Step 1: Map your CSR arrays into an Eigen::SparseMatrix
  typedef Eigen::SparseMatrix<float, Eigen::RowMajor, int> SpMat;
  SpMat A(n, n);

  std::vector<Triplet<float>> triplets;
  triplets.reserve(nnz_count);
  for (size_t i = 0; i < (size_t)n; i++) {
    for (int j = csrRowPtr[i]; j < csrRowPtr[i + 1]; j++) {
      triplets.push_back(Triplet<float>(i, csrColInd[j], csrVal[j]));
    }
  }
  A.setFromTriplets(triplets.begin(), triplets.end());

  // Step 2: Map infectious_vector as a dense matrix (column-major!)
  Map<const Matrix<float, Dynamic, Dynamic, ColMajor>> B(infectious_vector, n,
                                                         m);

  // Step 3: Map exposure_matrix as output
  Map<Matrix<float, Dynamic, Dynamic, ColMajor>> C(exposure_matrix, n, m);

  // Step 4: Sparse × Dense multiplication
  C = A * B;
}

/*
 * CPUComputeInfectionProbability
 *   - Replicates compute_infection_probability kernel
 */
void CPUComputeInfectionProbability(float *exposure_matrix,
                                    const int *countdown_vector, int n, int m,
                                    float step_size) {
  int total = n * m;
  for (int idx = 0; idx < total; idx++) {
    int row = idx % n;
    int col = idx / n;
    if (countdown_vector[row + col * n] == SUSCEPTIBLE) {
      float infect_prob = gSusceptibleInfectProb;
      float log_base = logf(1.0f - infect_prob);
      //  float e = exposure_matrix[idx] / step_size;
      //  we do not normalize wrt step size anymore - we now normalize wrt a
      //  standard "exposure hour". Just easier to work with.
      float e = exposure_matrix[row + col * n] / 3600.0f;
      exposure_matrix[idx] = 1.0f - expf(log_base * e);
    } else {
      exposure_matrix[idx] = 0.0f;
    }
  }
}

/*
 * CPUMonteCarloSimulation
 *   - Replicates monte_carlo_simulation kernel
 */
void CPUMonteCarloSimulation(const float *prob_matrix, int *countdown_vector,
                             int n, int m, float step_size) {
  // We'll do a simple rand() approach again.
  // You can seed it once per iteration for demonstration.
  srand((unsigned)time(NULL));
  int total = n * m;
  for (int idx = 0; idx < total; idx++) {
    int row = idx % n;
    int col = idx / n;

    if (countdown_vector[row + col * n] == SUSCEPTIBLE) {
      float rand_val = (float)rand() / (float)RAND_MAX;
      float infectionProb = prob_matrix[idx];
      if (infectionProb > rand_val) {
        // New infection => set to incubation
        countdown_vector[row + col * n] = gUpperRange + step_size;
      }
    }
  }
}

/*
 * CPUUpdateCountdownVector
 *   - Replicates update_countdown_vector kernel
 */
void CPUUpdateCountdownVector(int *countdown_vector, int n, int m,
                              float step_size) {
  int total = n * m;
  for (int idx = 0; idx < total; idx++) {
    if (countdown_vector[idx] > 0) {
      countdown_vector[idx] -= step_size;
    }
    if (countdown_vector[idx] < 0) {
      countdown_vector[idx] = SUSCEPTIBLE;
    }
  }
}
/* ----------------------------------------------------------------------------
 * END of CPU FALLBACK FUNCTIONS
 * ----------------------------------------------------------------------------*/

void write_simulation_state(FILE *summary_fp, FILE *node_fp,
                            const int *countdown_vector, int num_nodes, int gM,
                            int64_t timestamp) {
  for (int sim = 0; sim < gM; sim++) {
    int incubating = 0, infectious = 0, resistant = 0, susceptible = 0;

    for (int node = 0; node < num_nodes; node++) {
      int idx = node + sim * num_nodes;
      int state_val = countdown_vector[idx];

      const char *state_str;
      // if (state_val == SUSCEPTIBLE) {
      //   susceptible++;
      //   state_str = "susceptible";
      // } else if (state_val > gMediumRange) {
      //   incubating++;
      //   state_str = "incubating";
      // } else if (state_val > gLowerRange) {
      //   infectious++;
      //   state_str = "infectious";
      // } else {
      //   resistant++;
      //   state_str = "resistant";
      // }

      int elapsed = (state_val <= 0 || state_val > gUpperRange)
                ? 0 : (gUpperRange - state_val);

      if (state_val == SUSCEPTIBLE) {
          susceptible++;
          state_str = "susceptible";
      } else if (gExposedDuration > 0 && elapsed < gExposedDuration) {
          incubating++;
          state_str = "incubating";
      } else if (gInfectiousDuration > 0 &&
                elapsed < gExposedDuration + gInfectiousDuration) {
          infectious++;
          state_str = "infectious";
      } else if (gResistantDuration > 0 &&
                elapsed < gExposedDuration + gInfectiousDuration + gResistantDuration) {
          resistant++;
          state_str = "resistant";
      } else {
          susceptible++;
          state_str = "susceptible";
      }

      if (node_fp) {
        fprintf(node_fp, "%lld,%d,%d,%s\n", (long long)timestamp, sim, node,
                state_str);
      }
    }

    if (summary_fp) {
      fprintf(summary_fp, "%lld,%d,%d,%d,%d,%d\n", (long long)timestamp, sim,
              incubating, infectious, resistant, susceptible);
    }
  }

  if (summary_fp)
    fflush(summary_fp);
  if (node_fp)
    fflush(node_fp);
}

int main(int argc, char **argv) {
  printf("=== GPU-Accelerated Temporal Epidemiology Simulation ===\n");

  if (argc < 2) {
    fprintf(stderr, "Usage: %s <graph file> [options]\n", argv[0]);
    fprintf(stderr, "Options:\n");
    fprintf(stderr, "  --cpu-only             Run on CPU only.\n");
    fprintf(
        stderr,
        "  --cpu-threads <int>    Number of CPU threads Eigen should use.\n");
    fprintf(stderr,
            "  --N <int>              Number of individuals per simulation.\n");
    fprintf(stderr,
            "  --M <int>              Number of parallel simulations.\n");
    fprintf(stderr, "  --step-size <float>    Time step size.\n");
    fprintf(stderr,
            "  --iterations <int>     Number of main simulation steps.\n");
    fprintf(stderr,
            "  --summary-out <file>   Output file for summary state counts.\n");
    fprintf(stderr,
            "  --node-state-out <file> Output file for per-node states.\n");
    fprintf(stderr, "  --initial-infected <float>     Initial infection "
                    "probability [0–1]\n");
    fprintf(
        stderr,
        "  --infect-prob <float>          Infection prob if exposed [0–1]\n");
    // fprintf(stderr, "  --upper-range <int>            Countdown time for
    // resistant phase\n"); fprintf(stderr, "  --medium-range <int> Countdown
    // time for infectious phase\n"); fprintf(stderr, "  --lower-range <int>
    // Threshold for susceptible state\n");
    fprintf(stderr, "  --exposed-duration <int>       Duration (seconds) "
                    "before an infected person becomes infectious\n");
    fprintf(stderr, "  --infectious-duration <int>    Duration (seconds) a "
                    "person remains infectious before recovery\n");
    fprintf(stderr,
            "  --resistant-duration <int>     Duration (seconds) a recovered "
            "person remains resistant before becoming susceptible again\n");
    fprintf(
        stderr,
        "  --start-time <epoch>         Unix start time of the simulation.\n");
    fprintf(stderr, "  --static-network-duration <seconds>  Duration (seconds) of each static contact network window.\n");
    fprintf(stderr, "  [deprecated] --time-step <seconds>   Alias for --static-network-duration.\n");


    return 1;
  }

  const char *filename = NULL;
  bool cpu_only = false;

  // Simple argument parsing:
  for (int i = 1; i < argc; i++) {
    if (argv[i][0] != '-') {
      filename = argv[i];
      continue;
    }
    if (strcmp(argv[i], "--cpu-only") == 0) {
      cpu_only = true;
    } else if (strcmp(argv[i], "--cpu-threads") == 0 && (i + 1 < argc)) {
      gEigenThreads = atoi(argv[++i]);
    } else if (strcmp(argv[i], "--M") == 0 && (i + 1 < argc)) {
      gM = atoi(argv[++i]);
    } else if (strcmp(argv[i], "--step-size") == 0 && (i + 1 < argc)) {
      gStepSize = (float)atof(argv[++i]);
    } else if (strcmp(argv[i], "--iterations") == 0 && (i + 1 < argc)) {
      gIterations = atoi(argv[++i]);
    } else if (strcmp(argv[i], "--summary-out") == 0 && (i + 1 < argc)) {
      remove(argv[i + 1]); // delete existing file if present
      summary_fp = fopen(argv[++i], "w");
      if (!summary_fp) {
        fprintf(stderr, "Error: Could not open summary output file.\n");
        return 1;
      }
    } else if (strcmp(argv[i], "--node-state-out") == 0 && (i + 1 < argc)) {
      remove(argv[i + 1]); // delete existing file if present
      node_fp = fopen(argv[++i], "w");
      if (!node_fp) {
        fprintf(stderr, "Error: Could not open node state output file.\n");
        return 1;
      }
    } else if (strcmp(argv[i], "--initial-infected") == 0 && (i + 1 < argc)) {
      gInitialInfectedProb = (float)atof(argv[++i]);
    } else if (strcmp(argv[i], "--infect-prob") == 0 && (i + 1 < argc)) {
      gSusceptibleInfectProb = (float)atof(argv[++i]);
    }
    // else if (strcmp(argv[i], "--upper-range") == 0 && (i+1 < argc)) {
    //     gUpperRange = atoi(argv[++i]);
    // }
    // else if (strcmp(argv[i], "--medium-range") == 0 && (i+1 < argc)) {
    //     gMediumRange = atoi(argv[++i]);
    // }
    // else if (strcmp(argv[i], "--lower-range") == 0 && (i+1 < argc)) {
    //     gLowerRange = atoi(argv[++i]);
    // }
    else if (strcmp(argv[i], "--exposed-duration") == 0 && (i + 1 < argc)) {
      gExposedDuration = atoi(argv[++i]);
    } else if (strcmp(argv[i], "--infectious-duration") == 0 &&
               (i + 1 < argc)) {
      gInfectiousDuration = atoi(argv[++i]);
    } else if (strcmp(argv[i], "--resistant-duration") == 0 && (i + 1 < argc)) {
      gResistantDuration = atoi(argv[++i]);
    } else if (strcmp(argv[i], "--start-time") == 0 && (i + 1 < argc)) {
      gGlobalStartTime = (int64_t)atoll(argv[++i]);
    } else if ((strcmp(argv[i], "--static-network-duration") == 0 ||
          strcmp(argv[i], "--time-step") == 0) && (i + 1 < argc)) {
      // Accept both --static-network-duration and legacy --time-step
      gStaticNetworkDuration = (int64_t)atoll(argv[++i]);
    }
    else {
      fprintf(stderr, "Unrecognized option: %s\n", argv[i]);
      return 1;
    }
  }

  if (!filename) {
    fprintf(stderr, "Missing contact network file argument.\n");
    return 1;
  }

  // After file opening, write headers:
  if (summary_fp) {
    fprintf(summary_fp,
            "time,simulation_id,exposed,infectious,resistant,susceptible\n");
  }
  if (node_fp) {
    fprintf(node_fp, "time,simulation_id,node_id,state\n");
  }

  if (cpu_only) {
    if (gEigenThreads > 0) {
      Eigen::setNbThreads(gEigenThreads);
      printf("[INFO] Eigen thread count set to %d\n", Eigen::nbThreads());
    } else {
      // Let Eigen decide
      printf("[INFO] Eigen will auto-select thread count: %d\n",
             Eigen::nbThreads());
    }
  }

  // set countdown ranges based on specified durations
  gUpperRange = gExposedDuration + gInfectiousDuration + gResistantDuration;
  gMediumRange = gInfectiousDuration + gResistantDuration;
  gLowerRange = gResistantDuration;

  /* 1) Parse graph => intervals */
  int edge_count = 0;
  IntervalMap *intervals =
      parse_edgelist_build_intervals(filename, &edge_count);
  if (!intervals || edge_count == 0) {
    fprintf(stderr, "No intervals read or parse error.\n");
    free_node_map();
    return 1;
  }
  num_nodes = get_total_node_count();
  printf("[INFO] Node count set from NODE_LIST: %d\n", num_nodes);
  fprintf(stderr, "Parsed %d edges (intervals) from graph.\n", edge_count);

  /* 2) Build NCLS from intervals */
  IntervalDBWrapper dbw;
  if (!build_interval_db_wrapper(intervals, edge_count, &dbw)) {
    fprintf(stderr, "Error: Failed to build interval DB.\n");
    free_node_map();
    return 1;
  }
  fprintf(stderr, "NCLS built. n=%d, nlists=%d.\n", dbw.n, dbw.nlists);

  /* 3) Simulation Setup */
  size_t totalSize = (size_t)num_nodes * (size_t)gM;

  // We will keep host arrays for CPU or for debugging GPU results
  int *h_countdown_vector = NULL;
  float *h_exposure_matrix = NULL;
  float *h_infectious_vector = NULL;

  // Allocate host arrays
  // For CPU mode, we do everything in these arrays
  // For GPU mode, we also store the data on GPU, but keep these for debug
  h_countdown_vector = (int *)malloc(totalSize * sizeof(int));
  h_exposure_matrix = (float *)malloc(totalSize * sizeof(float));
  h_infectious_vector = (float *)malloc(totalSize * sizeof(float));
  if (!h_countdown_vector || !h_exposure_matrix || !h_infectious_vector) {
    fprintf(stderr, "Error: unable to allocate host arrays.\n");
    free_node_map();
    free_interval_db_wrapper(&dbw);
    return 1;
  }

  // If not CPU-only, allocate GPU buffers:
  int *d_countdown_vector = NULL;
  float *d_exposure_matrix = NULL;
  float *d_infectious_vector = NULL;

  if (!cpu_only) {
    CUDA_CHECK(
        hipMalloc((void **)&d_countdown_vector, totalSize * sizeof(int)));
    CUDA_CHECK(
        hipMalloc((void **)&d_exposure_matrix, totalSize * sizeof(float)));
    CUDA_CHECK(
        hipMalloc((void **)&d_infectious_vector, totalSize * sizeof(float)));
  }

  // Initialize countdown vector
  if (cpu_only) {
    CPUInitializeCountdownVector(h_countdown_vector, (int)totalSize,
                                 gInitialInfectedProb, gStepSize);
  } else {
    // GPU initialization
    int threads = 256;
    int blocks = (int)((totalSize + threads - 1) / threads);
    initialize_countdown_vector<<<blocks, threads>>>(
        d_countdown_vector, (int)totalSize, gInitialInfectedProb, gStepSize,
        (unsigned long)time(NULL), gMediumRange, SUSCEPTIBLE);
    CUDA_CHECK(hipDeviceSynchronize());
    // Copy to host for the initial debug
    CUDA_CHECK(hipMemcpy(h_countdown_vector, d_countdown_vector,
                          totalSize * sizeof(int), hipMemcpyDeviceToHost));
  }

  // Create cuSPARSE handle (only if not CPU-only)
  hipsparseHandle_t cusparseHandle = NULL;
  if (!cpu_only) {
    hipsparseCreate(&cusparseHandle);
  }

  // Prepare DnMat descriptors (only if not CPU-only)
  hipsparseDnMatDescr_t matB = NULL, matC = NULL;
  if (!cpu_only) {
    hipsparseCreateDnMat(&matB, (int64_t)num_nodes, (int64_t)gM,
                        (int64_t)num_nodes, d_infectious_vector, HIP_R_32F,
                        HIPSPARSE_ORDER_COL);
    hipsparseCreateDnMat(&matC, (int64_t)num_nodes, (int64_t)gM,
                        (int64_t)num_nodes, d_exposure_matrix, HIP_R_32F,
                        HIPSPARSE_ORDER_COL);
  }

  float alpha = 1.0f, beta = 0.0f;
  void *dBuffer = NULL;
  size_t bufferSize = 0;

  hipsparseSpMatDescr_t matA = NULL; // adjacency

  // write initial conditions to file
  if (summary_fp || node_fp) { // if file output flags exist
    write_simulation_state(summary_fp, node_fp, h_countdown_vector, num_nodes,
                           gM, gGlobalStartTime);
  }

  printf("\n"); // for formatting
  printf("Starting temporal Monte Carlo simulation for %d iterations...\n",
         gIterations);
  printf("[INFO] Simulation start time: %lld, time step: %.3f seconds, static network window: %lld seconds\n",
         (long long)gGlobalStartTime, (double)gStepSize,
         (long long)gStaticNetworkDuration);

  if (gStaticNetworkDuration <= 0) {
    fprintf(stderr, "Error: static network duration must be positive.\n");
    return 1;
  }

  // Basic GPU thread config for kernels (used if not CPU-only)
  int threads = 256;
  int blocks = (int)((totalSize + threads - 1) / threads);

  /* 4) Main iteration loop */
  for (int iter = 0; iter < gIterations; iter++) {
    double raw_offset = (double)iter * (double)gStepSize;
    if (raw_offset > (double)INT64_MAX || raw_offset < (double)INT64_MIN) {
      fprintf(stderr,
              "Error: Overflow detected when computing time offset.\n");
      return 1;
    }

    int64_t offset = (int64_t)llround(raw_offset);

    if ((offset > 0 && gGlobalStartTime > INT64_MAX - offset) ||
        (offset < 0 && gGlobalStartTime < INT64_MIN - offset)) {
      fprintf(stderr,
              "Error: Overflow detected when computing current simulation time.\n");
      return 1;
    }

    int64_t current_time_ts = gGlobalStartTime + offset;

    // Align network lookup to the static slice that contains the current
    // simulation time so several iterations can reuse the same contact window.
    int64_t network_offset = (offset / gStaticNetworkDuration) * gStaticNetworkDuration;
    if (offset < 0 && (offset % gStaticNetworkDuration)) {
      network_offset -= gStaticNetworkDuration;
    }

    if ((network_offset > 0 && gGlobalStartTime > INT64_MAX - network_offset) ||
        (network_offset < 0 && gGlobalStartTime < INT64_MIN - network_offset)) {
      fprintf(stderr,
              "Error: Overflow detected when computing network window start.\n");
      return 1;
    }

    int64_t current_start_ts = gGlobalStartTime + network_offset;
    if ((gStaticNetworkDuration > 0 &&
         current_start_ts > INT64_MAX - gStaticNetworkDuration) ||
        (gStaticNetworkDuration < 0 &&
         current_start_ts < INT64_MIN - gStaticNetworkDuration)) {
      fprintf(stderr,
              "Error: Overflow detected when computing current_end_ts.\n");
      return 1;
    }

    int64_t current_end_ts = current_start_ts + gStaticNetworkDuration;

    printf("\n--- Iteration %d (sim time %lld): time window [%lld, %lld) ---\n",
           iter + 1, (long long)current_time_ts, (long long)current_start_ts,
           (long long)current_end_ts);

    /*
     * Allocate buffer for overlapping intervals ("hits").
     * Normally, we use dbw.nlists (number of sublists) as the max expected
     * overlap count. However, if all intervals are top-level (i.e., no
     * nesting), dbw.nlists == 0, and using it would result in a zero-sized
     * buffer. In that case, we conservatively allocate up to dbw.n (total
     * number of intervals), which guarantees enough space.
     */
    IntervalMap *hits = (IntervalMap *)malloc(
        sizeof(IntervalMap) * (dbw.nlists > 0 ? dbw.nlists : dbw.n));

    if (!hits) {
      fprintf(stderr, "Out of memory for hits.\n");
      exit(1);
    }
    IntervalIterator *it = interval_iterator_alloc();
    if (!it) {
      fprintf(stderr, "Out of memory for IntervalIterator.\n");
      free(hits);
      exit(1);
    }
    int n_return = 0;

    find_intervals(it, current_start_ts, current_end_ts, dbw.im, dbw.n,
                   dbw.subheader, dbw.nlists, hits, dbw.nlists, &n_return, &it);

    free_interval_iterator(it);

    /* 4.2) Convert the returned intervals to CSR adjacency. */
    int *h_csrRowPtr = NULL;
    int *h_csrColInd = NULL;
    float *h_csrVal = NULL;
    size_t nnz_count = 0;

    build_csr_from_intervals(hits, n_return, num_nodes, &h_csrRowPtr,
                             &h_csrColInd, &h_csrVal, &nnz_count,
                             current_start_ts, current_end_ts);

    free(hits);

    printf("  iteration %d => found %d intervals => nnz=%zu\n", iter + 1,
           n_return, nnz_count);

    // If CPU-only, we won't copy to device; we'll do a CPU spMM
    int *d_csrRowPtr = NULL;
    int *d_csrColInd = NULL;
    float *d_csrVal = NULL;

    if (!cpu_only) {
      /* 4.3) Copy CSR to GPU memory. */
      CUDA_CHECK(
          hipMalloc((void **)&d_csrRowPtr, (num_nodes + 1) * sizeof(int)));
      CUDA_CHECK(hipMalloc((void **)&d_csrColInd, nnz_count * sizeof(int)));
      CUDA_CHECK(hipMalloc((void **)&d_csrVal, nnz_count * sizeof(float)));

      CUDA_CHECK(hipMemcpy(d_csrRowPtr, h_csrRowPtr,
                            (num_nodes + 1) * sizeof(int),
                            hipMemcpyHostToDevice));
      CUDA_CHECK(hipMemcpy(d_csrColInd, h_csrColInd, nnz_count * sizeof(int),
                            hipMemcpyHostToDevice));
      CUDA_CHECK(hipMemcpy(d_csrVal, h_csrVal, nnz_count * sizeof(float),
                            hipMemcpyHostToDevice));

      /* 4.4) Create/Update the cuSPARSE SpMat descriptor for A. */
      if (matA) {
        hipsparseDestroySpMat(matA);
        matA = NULL;
      }
      hipsparseCreateCsr(&matA, (int64_t)num_nodes, (int64_t)num_nodes,
                        (int64_t)nnz_count, d_csrRowPtr, d_csrColInd, d_csrVal,
                        HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                        HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F);

      /* 4.5) We now must get buffer size for SpMM with this adjacency. */
      hipsparseSpMM_bufferSize(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                              HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, matA,
                              matB, &beta, matC, HIP_R_32F,
                              HIPSPARSE_SPMM_ALG_DEFAULT, &bufferSize);

      if (dBuffer) {
        hipFree(dBuffer);
        dBuffer = NULL;
      }
      CUDA_CHECK(hipMalloc(&dBuffer, bufferSize));
    }

    /* 4.6) Steps: Infectious vector, SpMM => exposure, infection prob, etc. */

    if (!cpu_only) {
      // GPU-based approach
      // 4.6a: generate_infectious_vector
      generate_infectious_vector<<<blocks, threads>>>(
          d_countdown_vector, d_infectious_vector, num_nodes, gM, gLowerRange,
          gMediumRange);
      CUDA_CHECK(hipDeviceSynchronize());

      // 4.6b: SpMM => exposure_matrix = A * infectious_vector
      hipsparseSpMM(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                   HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, matA, matB, &beta,
                   matC, HIP_R_32F, HIPSPARSE_SPMM_ALG_DEFAULT, dBuffer);
      CUDA_CHECK(hipDeviceSynchronize());

      // 4.6c: compute_infection_probability
      compute_infection_probability<<<blocks, threads>>>(
          d_exposure_matrix, d_countdown_vector, num_nodes, gM, gStepSize,
          gSusceptibleInfectProb, SUSCEPTIBLE);
      CUDA_CHECK(hipDeviceSynchronize());

      // 4.6d: monte_carlo_simulation
      monte_carlo_simulation<<<blocks, threads>>>(
          d_exposure_matrix, d_countdown_vector, num_nodes, gM,
          (unsigned long)(time(NULL) + iter), gStepSize, gUpperRange,
          SUSCEPTIBLE);

      CUDA_CHECK(hipDeviceSynchronize());

      // 4.6e: update_countdown_vector
      update_countdown_vector<<<blocks, threads>>>(d_countdown_vector,
                                                   num_nodes, gM, gStepSize);
      CUDA_CHECK(hipDeviceSynchronize());

      // For debug printing
      CUDA_CHECK(hipMemcpy(h_countdown_vector, d_countdown_vector,
                            totalSize * sizeof(int), hipMemcpyDeviceToHost));
      print_status_counts(h_countdown_vector, num_nodes, gM, 5);

      if (summary_fp || node_fp) { // if file output flags set
        write_simulation_state(summary_fp, node_fp, h_countdown_vector,
                               num_nodes, gM, current_time_ts);
      }

      /* 4.7) Cleanup adjacency from GPU */
      if (matA) {
        hipsparseDestroySpMat(matA);
        matA = NULL;
      }
      CUDA_CHECK(hipFree(d_csrRowPtr));
      CUDA_CHECK(hipFree(d_csrColInd));
      CUDA_CHECK(hipFree(d_csrVal));
      d_csrRowPtr = NULL;
      d_csrColInd = NULL;
      d_csrVal = NULL;
    } else {
      // CPU-only approach
      // 4.6a: generate_infectious_vector
      CPUGenerateInfectiousVector(h_countdown_vector, h_infectious_vector,
                                  num_nodes, gM);

      // 4.6b: SpMM => exposure_matrix = A * infectious_vector
      // (we have adjacency in h_csrRowPtr, h_csrColInd, h_csrVal)
      CPUComputeSpMM(h_csrRowPtr, h_csrColInd, h_csrVal, nnz_count, num_nodes,
                     h_infectious_vector, // NxM
                     h_exposure_matrix,   // NxM
                     gM);

      // 4.6c: compute_infection_probability
      CPUComputeInfectionProbability(h_exposure_matrix, h_countdown_vector,
                                     num_nodes, gM, gStepSize);

      // 4.6d: monte_carlo_simulation
      CPUMonteCarloSimulation(h_exposure_matrix, h_countdown_vector, num_nodes,
                              gM, gStepSize);

      // 4.6e: update_countdown_vector
      CPUUpdateCountdownVector(h_countdown_vector, num_nodes, gM, gStepSize);

      // Debug print first 5 "rows"
      print_status_counts(h_countdown_vector, num_nodes, gM, 5);

      if (summary_fp || node_fp) {
        write_simulation_state(summary_fp, node_fp, h_countdown_vector,
                               num_nodes, gM, current_time_ts);
      }

      // CPU adjacency arrays can just be freed
    }

    // 4.7) Cleanup adjacency from CPU side
    free(h_csrRowPtr);
    free(h_csrColInd);
    free(h_csrVal);
  }

  printf("Simulation done.\n");

  /* 5) Cleanup GPU objects if used */
  if (!cpu_only) {
    if (dBuffer)
      hipFree(dBuffer);
    if (matA)
      hipsparseDestroySpMat(matA);
    hipsparseDestroyDnMat(matB);
    hipsparseDestroyDnMat(matC);
    hipsparseDestroy(cusparseHandle);

    CUDA_CHECK(hipFree(d_countdown_vector));
    CUDA_CHECK(hipFree(d_exposure_matrix));
    CUDA_CHECK(hipFree(d_infectious_vector));
  }

  // Close files
  if (summary_fp)
    fclose(summary_fp);
  if (node_fp)
    fclose(node_fp);

  // Free host arrays
  free(h_countdown_vector);
  free(h_exposure_matrix);
  free(h_infectious_vector);

  /* 6) Cleanup NCLS + node map */
  free_interval_db_wrapper(&dbw);
  free_node_map();

  return 0;
}
